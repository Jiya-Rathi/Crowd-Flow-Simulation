#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void matmul_kernel(float* A, float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0;
        for (int i = 0; i < K; ++i)
            sum += A[row * K + i] * B[i * N + col];
        C[row * N + col] = sum;
    }
}

void matmul(torch::Tensor A, torch::Tensor B, torch::Tensor C, int M, int N, int K) {
    dim3 threads(16, 16);
    dim3 blocks((N+15)/16, (M+15)/16);
    matmul_kernel<<<blocks, threads>>>(
        A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), M, N, K
    );
}